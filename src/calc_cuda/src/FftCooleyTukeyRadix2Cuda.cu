#ifndef BIT_REVERSE_SHIFT_VALUE
# define BIT_REVERSE_SHIFT_VALUE 26
#endif

#include <builtin_types.h>
#include <hip/hip_runtime.h>

#include <complex>

__global__ void bit_reverse_permutation(
    const float2* input,
          float2* output
) {
    unsigned int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int i1 = globalId;
    unsigned int i2 = globalId;

    i2 = ((i2 >> 1) & 0x55555555) | ((i2 & 0x55555555) << 1);
    i2 = ((i2 >> 2) & 0x33333333) | ((i2 & 0x33333333) << 2);
    i2 = ((i2 >> 4) & 0x0F0F0F0F) | ((i2 & 0x0F0F0F0F) << 4);
    i2 = ((i2 >> 8) & 0x00FF00FF) | ((i2 & 0x00FF00FF) << 8);
    i2 = (i2 >> 16) | (i2 << 16);

    i2 = i2 >> BIT_REVERSE_SHIFT_VALUE;

    output[i2] = input[i1];
}

void bit_reverse_permutation_wrapper(const float* input,
                                           float* output,
                                           size_t size,
                                           size_t block_size) {
    float2* in, *out;

    hipMalloc((void**)&in,  sizeof(float2) * size);
    hipMalloc((void**)&out, sizeof(float2) * size);

    hipMemcpy(in, input, sizeof(float2) * size, hipMemcpyHostToDevice);

    bit_reverse_permutation<<<block_size, size / block_size>>>(in, out);

    hipDeviceSynchronize();

    hipMemcpy(output, out, sizeof(float2) * size, hipMemcpyDeviceToHost);

    hipFree(in);
    hipFree(out);
}

__global__ void fft_stage(
    const float2* input,
          float2* output,
    const float2* omegaValues,
    unsigned int subFftSize,
    unsigned int subFftCount,
    unsigned int stageIndex
) {
    const unsigned int subFftIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int subFftElementIndex = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int subFftHalfSize = subFftSize / 2;

    const unsigned int index1 = subFftIndex * subFftSize + subFftElementIndex;
    const unsigned int index2 = index1 + subFftHalfSize;

    const float2 input1 = input[index1];
    const float2 input2 = input[index2];

    const float2 omegaK = omegaValues[subFftElementIndex];
    float2 mul;

    mul.x = omegaK.x * input2.x - omegaK.y * input2.y;
    mul.y = omegaK.x * input2.y + omegaK.y * input2.x;

    float2 y1 = input1;
    float2 y2 = input1;

    y1.x += mul.x;
    y1.y += mul.y;
    y2.x -= mul.x;
    y2.y -= mul.y;

    output[index1] = y1;
    output[index2] = y2;
}

__global__ void calculate_magnitudes(
    const float2* fft,
    float* magnitudes
) {
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    magnitudes[i] = 2 * sqrt((float)(pow(fft[i].x, 2) + pow(fft[i].y, 2)));
}

void calculate_magnitudes_wrapper(const float* fft,
                                        float* magnitudes,
                                        size_t size,
                                        size_t block_size) {
    float2* in;
    float* out;

    hipMalloc((void**)&in,  sizeof(float2) * size);
    hipMalloc((void**)&out, sizeof(float ) * size);

    hipMemcpy(in, fft, sizeof(float2) * size, hipMemcpyHostToDevice);

    calculate_magnitudes<<<block_size, size / block_size>>>(in, out);

    hipDeviceSynchronize();

    hipMemcpy(magnitudes, out, sizeof(float) * size, hipMemcpyDeviceToHost);

    hipFree(in);
    hipFree(out);
}


__global__ void find_max(
    const float* values,
    float* temp,
    float* output
) {
    const size_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t localId = threadIdx.x;
    const size_t localSize = blockDim.x;

    temp[localId] = values[globalId];
    // barrier(CLK_LOCAL_MEM_FENCE);

    for (int i = localSize >> 1; i > 0; i >>= 1)
    {
        if (localId < i)
        {
            if (temp[localId] < temp[localId + i])
            {
                temp[localId] = temp[localId + i];
            }

            // barrier(CLK_LOCAL_MEM_FENCE);
        }
    }

    if (localId == 0)
    {
        output[blockIdx.x] = temp[0];
    }
}

void find_max_wrapper(const float* values,
                            float* output,
                            size_t size,
                            size_t block_size) {
    float* in, *tmp, *out;

    hipMalloc((void**)&in,  sizeof(float) * size);
    hipMalloc((void**)&tmp, sizeof(float) * size);
    hipMalloc((void**)&out, sizeof(float) * size);

    hipMemcpy(in, values, sizeof(float) * size, hipMemcpyHostToDevice);

    find_max<<<block_size, size / block_size>>>(in, tmp, out);

    hipDeviceSynchronize();

    hipMemcpy(output, out, sizeof(float) * size, hipMemcpyDeviceToHost);

    hipFree(in);
    hipFree(tmp);
    hipFree(out);
}

// todo: don't copy each time.
void fft_stage_wrapper(const float* input,
                             float* output,
                       const float* omegaValues,
                       unsigned int subFftSize,
                       unsigned int subFftCount,
                       unsigned int stageIndex,
                       size_t input_size,
                       size_t omega_size) {
    float2* in, *out, *omega;

    dim3 blocks(subFftCount, subFftSize / 2, 1);
    dim3 threads(1, 1, 1);

    hipMalloc((void**)&in,    sizeof(float2) * input_size);
    hipMalloc((void**)&out,   sizeof(float2) * input_size);
    hipMalloc((void**)&omega, sizeof(float2) * omega_size);

    hipMemcpy(in,    input,       sizeof(float2) * input_size, hipMemcpyHostToDevice);
    hipMemcpy(omega, omegaValues, sizeof(float2) * omega_size, hipMemcpyHostToDevice);

    fft_stage<<<blocks, threads>>>(in, out, omega, subFftSize, subFftCount, stageIndex);

    hipDeviceSynchronize();

    hipMemcpy(output, out, sizeof(float2) * input_size, hipMemcpyDeviceToHost);

    hipFree(in);
    hipFree(out);
    hipFree(omega);
}
