#include "hip/hip_runtime.h"
#ifndef BIT_REVERSE_SHIFT_VALUE
# define BIT_REVERSE_SHIFT_VALUE 26
#endif

#include <builtin_types.h>
#include <hip/hip_runtime.h>
#include <>
#include <complex>

unsigned int bitReverse(unsigned int v)
{
    v = ((v >> 1) & 0x55555555) | ((v & 0x55555555) << 1);
    v = ((v >> 2) & 0x33333333) | ((v & 0x33333333) << 2);
    v = ((v >> 4) & 0x0F0F0F0F) | ((v & 0x0F0F0F0F) << 4);
    v = ((v >> 8) & 0x00FF00FF) | ((v & 0x00FF00FF) << 8);
    v = (v >> 16) | (v << 16);

    v = v >> BIT_REVERSE_SHIFT_VALUE;

    return v;
}

float2 complexMultiply(float2 a, float2 b) {
    return (float2)(a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x);
}

__global__ void bit_reverse_permutation(
    const float2* input,
    float2* output
) {
    unsigned int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int i1 = globalId;
    unsigned int i2 = bitReverse(globalId);
    output[i2] = input[i1];
}

__global__ void fft_stage(
    const float2* input,
    float2* output,
    const float2* omegaValues,
    unsigned int subFftSize,
    unsigned int subFftCount,
    unsigned int stageIndex
) {
    const unsigned int subFftIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int subFftElementIndex = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int subFftHalfSize = subFftSize / 2;

    const unsigned int index1 = subFftIndex * subFftSize + subFftElementIndex;
    const unsigned int index2 = index1 + subFftHalfSize;

    const float2 input1 = input[index1];
    const float2 input2 = input[index2];

    const float2 omegaK = omegaValues[subFftElementIndex];
    const float2 mul = complexMultiply(omegaK, input2);
    float2 y1 = input1;
    float2 y2 = input1;

    y1.x += mul.x;
    y1.y += mul.y;
    y2.x -= mul.x;
    y2.y -= mul.y;

    output[index1] = y1;
    output[index2] = y2;
}

__global__ void calculate_magnitudes(
    const float2* fft,
    float* magnitudes
) {
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    magnitudes[i] = 2 * sqrt((float)(pow(fft[i].x, 2) + pow(fft[i].y, 2)));
}

__global__ void find_max(
    const float* values,
    __shared__ float* temp,
    float* output
) {
    const size_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t localId = threadIdx.x;
    const size_t localSize = blockDim.x;

    temp[localId] = values[globalId];

    for (int i = localSize >> 1; i > 0; i >>= 1)
    {
        if (localId < i)
        {
            if (temp[localId] < temp[localId + i])
            {
                temp[localId] = temp[localId + i];
            }
        }
    }

    if (localId == 0)
    {
        output[blockIdx.x] = temp[0];
    }
}
