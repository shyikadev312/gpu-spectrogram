#include <builtin_types.h>
#include <hip/hip_runtime.h>

#include <complex>
__global__ void convertToDBFS(
    float* magnitudes,
    float referenceValue
) {
    unsigned int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    float magnitude = magnitudes[globalId];
    float dbfs = 20.0f * log10(magnitude / referenceValue);
    if (dbfs < -10000.0f)
    {
        dbfs = -10000.0f;
    }
    magnitudes[globalId] = dbfs;
}

__global__ void updateDensityHeatmap(
    float2* heatmap,
    float* dbfsHistoryBuffer,
    unsigned int heatmapWidth,
    unsigned int heatmapHeight,
    unsigned int historyBufferCount,
    unsigned int mostRecentBufferIndex,
    float magnitudeIndexToDbfsCoeff
) {
    const unsigned int frequencyIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int magnitudeCellIndex = blockIdx.y * blockDim.y + threadIdx.y;

    const float currentCellCorrespondingDBFS = -((float)magnitudeCellIndex) * magnitudeIndexToDbfsCoeff;

    float cellValue = 0;

    float dbfsAge = 0;
    float dbfsIntensity = 0;

    for (size_t historyBufferIndex = 0; historyBufferIndex < historyBufferCount; ++historyBufferIndex)
    {
        const size_t historyBufferElementIndex = historyBufferIndex * heatmapWidth + frequencyIndex;
        const float dbfsValue = dbfsHistoryBuffer[historyBufferElementIndex];

        bool isPresent = (currentCellCorrespondingDBFS * 0.95f > dbfsValue) && (currentCellCorrespondingDBFS * 1.05f < dbfsValue);
        if (isPresent)
        {
            cellValue += 1.0f;
            dbfsIntensity += 1.0f;

            float bufferAge = 0;
            if (mostRecentBufferIndex >= historyBufferIndex)
            {
                bufferAge = mostRecentBufferIndex - historyBufferIndex;
            } else
            {
                bufferAge = historyBufferCount - (historyBufferIndex - mostRecentBufferIndex);
            }

            dbfsAge += bufferAge;
        }
    }

    unsigned int heatmapCellIndex = frequencyIndex * heatmapHeight + magnitudeCellIndex;
    heatmap[heatmapCellIndex].x = dbfsIntensity / (float)historyBufferCount;
    heatmap[heatmapCellIndex].y = (dbfsAge / (float)historyBufferCount) / (float)historyBufferCount;
}
