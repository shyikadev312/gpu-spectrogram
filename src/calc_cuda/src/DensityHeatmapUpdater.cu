#include <builtin_types.h>
#include <hip/hip_runtime.h>

#include <complex>
__global__ void convert_to_dbfs(
    float* magnitudes,
    float referenceValue
) {
    unsigned int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    float magnitude = magnitudes[globalId];
    float dbfs = 20.0f * log10(magnitude / referenceValue);
    if (dbfs < -10000.0f)
    {
        dbfs = -10000.0f;
    }
    magnitudes[globalId] = dbfs;
}

void convert_to_dbfs_wrapper(float* magnitudes,
                           float referenceValue,
                           size_t size) {
    float* mag;

    hipMalloc((void**)&mag, sizeof(float) * size);

    hipMemcpy(mag, magnitudes, sizeof(float) * size, hipMemcpyHostToDevice);

    convert_to_dbfs<<<1, size>>>(mag, referenceValue);

    hipDeviceSynchronize();

    hipMemcpy(magnitudes, mag, sizeof(float) * size, hipMemcpyDeviceToHost);

    hipFree(mag);
}

__global__ void update_density_heatmap(
    float2* heatmap,
    float* dbfsHistoryBuffer,
    unsigned int heatmapWidth,
    unsigned int heatmapHeight,
    unsigned int historyBufferCount,
    unsigned int mostRecentBufferIndex,
    float magnitudeIndexToDbfsCoeff
) {
    const unsigned int frequencyIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int magnitudeCellIndex = blockIdx.y * blockDim.y + threadIdx.y;

    const float currentCellCorrespondingDBFS = -((float)magnitudeCellIndex) * magnitudeIndexToDbfsCoeff;

    float cellValue = 0;

    float dbfsAge = 0;
    float dbfsIntensity = 0;

    for (size_t historyBufferIndex = 0; historyBufferIndex < historyBufferCount; ++historyBufferIndex)
    {
        const size_t historyBufferElementIndex = historyBufferIndex * heatmapWidth + frequencyIndex;
        const float dbfsValue = dbfsHistoryBuffer[historyBufferElementIndex];

        bool isPresent = (currentCellCorrespondingDBFS * 0.95f > dbfsValue) && (currentCellCorrespondingDBFS * 1.05f < dbfsValue);
        if (isPresent)
        {
            cellValue += 1.0f;
            dbfsIntensity += 1.0f;

            float bufferAge = 0;
            if (mostRecentBufferIndex >= historyBufferIndex)
            {
                bufferAge = mostRecentBufferIndex - historyBufferIndex;
            } else
            {
                bufferAge = historyBufferCount - (historyBufferIndex - mostRecentBufferIndex);
            }

            dbfsAge += bufferAge;
        }
    }

    unsigned int heatmapCellIndex = frequencyIndex * heatmapHeight + magnitudeCellIndex;
    heatmap[heatmapCellIndex].x = dbfsIntensity / (float)historyBufferCount;
    heatmap[heatmapCellIndex].y = (dbfsAge / (float)historyBufferCount) / (float)historyBufferCount;
}

void update_density_heatmap_wrapper(float* heatmap,
                                  float* dbfsHistoryBuffer,
                                  unsigned int heatmapWidth,
                                  unsigned int heatmapHeight,
                                  unsigned int historyBufferCount,
                                  unsigned int mostRecentBufferIndex,
                                  float magnitudeIndexToDbfsCoeff,
                                  size_t size,
                                  size_t historyBufferSize) {
    float2* heat;
    float* history;

    hipMalloc((void**)&heat, sizeof(float2) * size);
    hipMalloc((void**)&history, sizeof(float) * historyBufferSize);

    hipMemcpy(heat, heatmap, sizeof(float2) * size, hipMemcpyHostToDevice);
    hipMemcpy(history, dbfsHistoryBuffer, sizeof(float) * historyBufferSize, hipMemcpyHostToDevice);

    update_density_heatmap <<<1, size >>>(heat, history, heatmapWidth, heatmapHeight, historyBufferCount, mostRecentBufferIndex, magnitudeIndexToDbfsCoeff);

    hipDeviceSynchronize();

    hipMemcpy(dbfsHistoryBuffer, history, sizeof(float) * historyBufferSize, hipMemcpyDeviceToHost);

    hipFree(heat);
    hipFree(history);
}
